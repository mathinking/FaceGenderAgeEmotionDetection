//
// File: main.cu
//
// GPU Coder version                    : 1.4
// CUDA/C/C++ source code generated on  : 07-Aug-2019 18:01:58
//

//***********************************************************************
// This automatically generated example CUDA main file shows how to call
// entry-point functions that MATLAB Coder generated. You must customize
// this file for your application. Do not modify this file directly.
// Instead, make a copy of this file, modify it, and integrate it into
// your development environment.
//
// This file initializes entry-point function arguments to a default
// size and value before calling the entry-point functions. It does
// not store or use any values returned from the entry-point functions.
// If necessary, it does pre-allocate memory for returned values.
// You can use this file as a starting point for a main function that
// you can deploy in your application.
//
// After you copy the file, and before you deploy it, you must make the
// following changes:
// * For variable-size function arguments, change the example sizes to
// the sizes that your application requires.
// * Change the example values of function arguments to the values that
// your application requires.
// * If the entry-point functions return values, store these values or
// otherwise use them as required by your application.
//
//***********************************************************************

// Include Files
#include "main.h"
#include <string.h>
#include <iostream>
#include "faceGenderAgeEmotionDetectionOnJetson.h"
#include "faceGenderAgeEmotionDetectionOnJetson_initialize.h"
#include "faceGenderAgeEmotionDetectionOnJetson_terminate.h"

// Function Declarations
static int main_faceGenderAgeEmotionDetectionOnJetson();

//
// Arguments    : void
// Return Type  : void
//
static int main_faceGenderAgeEmotionDetectionOnJetson(int32_T argc, const char * const argv[])
{
  // Initialize function 'faceGenderAgeEmotionDetectionOnJetson' input arguments.
  // Call the entry-point 'faceGenderAgeEmotionDetectionOnJetson'.
  
  bool bAge = false;
  bool bEmotion = false;
  bool bGender = false;
  
  switch(argc) {
    case 1: break;
    case 2: bGender = !bool(strcmp(argv[1],"true"));
            break;
    case 3: bGender = !bool(strcmp(argv[1],"true"));
            bAge = !bool(strcmp(argv[2],"true"));
            break;
    case 4: bGender = !bool(strcmp(argv[1],"true"));
            bAge = !bool(strcmp(argv[2],"true"));
            bEmotion = !bool(strcmp(argv[3],"true"));
            break;
    default: // Tell the user how to run the program
            std::cerr << "Usage: " << argv[0] << " boolean_Age boolean_Gender boolean_Emotion" <<std::endl<< "e.g.: "<< argv[0] << " true false false" << std::endl;
            return 1;
  }
  
  faceGenderAgeEmotionDetectionOnJetson(bGender, bAge, bEmotion);
}

//
// Arguments    : int32_T argc
//                const char * const argv[]
// Return Type  : int32_T
//
int32_T main(int32_T argc, const char * const argv[])
{
  // Initialize the application.
  // You do not need to do this more than one time.
  faceGenderAgeEmotionDetectionOnJetson_initialize();
  
  // Invoke the entry-point functions.
  // You can call entry-point functions multiple times.
  main_faceGenderAgeEmotionDetectionOnJetson(argc, argv);
  
  // Terminate the application.
  // You do not need to do this more than one time.
  faceGenderAgeEmotionDetectionOnJetson_terminate();
  return 0;
}

//
// File trailer for main.cu
//
// [EOF]
//
